#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#define TX 16
#define TY 32

__global__
void doCopyKernel(float *d_colornew,float *d_color, int colsize, int rowsize)
{
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    int x = Row*colsize+Col;

    if(Col<colsize && Row<rowsize)
        d_color[x] = d_colornew[x];

}
__global__ 
void performUpdatesKernel(float *d_colornew,float *d_color, int colsize, int rowsize)
{
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    int x = Row*colsize+Col;
    int xm = x-colsize;
    int xp = x+colsize;

    if(Col<colsize && Row<rowsize){

		if (Row != 0 && Row != (rowsize-1) && Col != 0 && Col != (colsize-1)){
			d_colornew[x] = (d_color[xp]+d_color[xm]+d_color[x+1]+d_color[x-1])/4;				   
		}
		else if (Row == 0 && Col != 0 && Col != (colsize-1)){
			d_colornew[x] = (d_color[xp]+d_color[x+1]+d_color[x-1])/3;
		}
		else if (Row == (rowsize-1) && Col != 0 && Col != (colsize-1)){
			d_colornew[x] = (d_color[xm]+d_color[x+1]+d_color[x-1])/3;
		}
		else if (Col == 0 && Row != 0 && Row != (rowsize-1)){
			d_colornew[x] = (d_color[xm]+d_color[xp]+d_color[x+1])/3;
		}
		else if (Col == (colsize-1) && Row != 0 && Row != (rowsize-1)){
			d_colornew[x] = (d_color[xm]+d_color[xp]+d_color[x-1])/3;
		}
		else if (Row==0 &&Col==0){
			d_colornew[x] = (d_color[x+1]+d_color[xp])/2;
		}
		else if (Row==0 &&Col==(colsize-1)){
			d_colornew[x] = (d_color[x-1]+d_color[xp])/2;
		}
		else if (Row==(rowsize-1) &&Col==0){
			d_colornew[x] = (d_color[x+1]+d_color[xm])/2;
		}
		else if (Row==(rowsize-1) &&Col==(colsize-1)){
			d_colornew[x] = (d_color[x-1]+d_color[xm])/2;
		}
	}
}
void performUpdates(float *h_colornew, float *h_color, int colsize, int rowsize, int nblurs)
{
    float *d_color,*d_colornew;
    int k;
    struct timeval tim;

    int sizef = sizeof(int)*colsize*rowsize;


    gettimeofday(&tim, NULL);
	double memaloc1=tim.tv_sec+(tim.tv_usec/1000000.0);

    hipMalloc((void **)&d_colornew,sizef);
    hipMalloc((void **)&d_color,sizef);

    gettimeofday(&tim, NULL);
	double memaloc2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Allocation of device memory took %.6lf seconds\n", memaloc2-memaloc1);

    gettimeofday(&tim, NULL);
	double host2dev1=tim.tv_sec+(tim.tv_usec/1000000.0);

    hipMemcpy(d_color,h_color,sizef,hipMemcpyHostToDevice);

    gettimeofday(&tim, NULL);
	double host2dev2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Host to device transfer took %.6lf seconds\n", host2dev2-host2dev1);

    dim3 dimGrid(ceil(colsize/(float)TX),ceil(rowsize/(float)TY),1);
    dim3 dimBlock(TX,TY,1);

    gettimeofday(&tim, NULL);
	double bluring1=tim.tv_sec+(tim.tv_usec/1000000.0);
    for(k=0;k<nblurs;++k){
        performUpdatesKernel<<<dimGrid,dimBlock>>>(d_colornew,d_color,colsize,rowsize);
        doCopyKernel<<<dimGrid,dimBlock>>>(d_colornew,d_color,colsize,rowsize);
    }
    gettimeofday(&tim, NULL);
	double bluring2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Bluring took %.6lf seconds\n", bluring2-bluring1);


    hipDeviceSynchronize();

    gettimeofday(&tim, NULL);
	double dev2host1=tim.tv_sec+(tim.tv_usec/1000000.0);

    hipMemcpy(h_colornew,d_color,sizef,hipMemcpyDeviceToHost);


    gettimeofday(&tim, NULL);
	double dev2host2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Device to Host transfer took %.6lf seconds\n", dev2host2-dev2host1);

    hipFree(d_colornew); hipFree(d_color);
}





int main (int argc, char *argv[])
{
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;

	gettimeofday(&tim, NULL);
	double inputfile1=tim.tv_sec+(tim.tv_usec/1000000.0);
	fp = fopen("David.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);
				
				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
	gettimeofday(&tim, NULL);
	double inputfile2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Reading the input file took %.6lf seconds\n", inputfile2-inputfile1);


	nblurs = 10;

	float *h_Rnew, *h_R, *h_Gnew, *h_G, *h_Bnew, *h_B;

	int nsize1=sizeof(float)*colsize*rowsize;


	h_Rnew = (float *)malloc(nsize1);
    h_R = (float *)malloc(nsize1);

    h_Gnew = (float *)malloc(nsize1);
    h_G = (float *)malloc(nsize1);

    h_Bnew = (float *)malloc(nsize1);
    h_B = (float *)malloc(nsize1);

    for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			h_R[row*colsize+col] = R[row][col];
    		h_G[row*colsize+col] = G[row][col];
    		h_B[row*colsize+col] = B[row][col];
    	}
    }
    

	performUpdates(h_Rnew,h_R,colsize,rowsize,nblurs);
	performUpdates(h_Gnew,h_G,colsize,rowsize,nblurs);
	performUpdates(h_Bnew,h_B,colsize,rowsize,nblurs);

	

	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			R[row][col]=h_Rnew[row*colsize+col];
    		G[row][col]=h_Gnew[row*colsize+col];
    		B[row][col]=h_Bnew[row*colsize+col];
    	}
    }



    gettimeofday(&tim, NULL);
	double outputfile1=tim.tv_sec+(tim.tv_usec/1000000.0);
	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
	gettimeofday(&tim, NULL);
	double outputfile2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf(" Reading the output file took %.6lf seconds\n", outputfile2-outputfile1);
    return 0;	
}